/*
 * SPDX-FileCopyrightText: Copyright (c) 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

static constexpr int threads_per_block = 256;
static constexpr size_t shared_mem_size = threads_per_block * sizeof(uint32_t);

__global__ void cuda_compare_checksum_kernel(const uint8_t** __restrict__ packet_ptrs,
                                             const size_t* __restrict__ packet_lengths,
                                             const uint32_t* __restrict__ expected_checksums,
                                             unsigned int* __restrict__ mismatch_counter,
                                             const int num_packets)
{
    // Each block will process one packet
    // while each thread will process a portion of the packet
    int packet_idx = blockIdx.x;

    if (packet_idx < num_packets) {
        extern __shared__ uint32_t shared_data[];

        // Get packet information
        const uint8_t* data = packet_ptrs[packet_idx];
        const size_t length = packet_lengths[packet_idx];

        uint32_t partial_sum = 0;
        unsigned int thread_id = threadIdx.x;
        unsigned int num_threads = blockDim.x;

        for (size_t i = thread_id; i < length; i += num_threads) {
            partial_sum += data[i];
        }

        shared_data[thread_id] = partial_sum;
        __syncthreads();

        // Reduce the sums of all blocks.
        for (unsigned int size = blockDim.x / 2; size > 0; size >>= 1) {
            if (thread_id < size) {
                shared_data[thread_id] += shared_data[thread_id + size];
            }
            __syncthreads();
        }

        // The first thread checks the checksum
        if (thread_id == 0) {
            const uint32_t computed_checksum = shared_data[0];
            const uint32_t expected_checksum = expected_checksums[packet_idx];

            if (computed_checksum != expected_checksum) {
                atomicAdd(mismatch_counter, 1);
            }
        }
    }
}

extern "C" void cuda_compare_checksum(const uint8_t** data_ptrs, const size_t* sizes,
                                      const uint32_t* expected_checksums,
                                      uint32_t* mismatch_counter, uint32_t num_packet)
{
    cuda_compare_checksum_kernel<<<num_packet, threads_per_block, shared_mem_size>>>(
        data_ptrs, sizes, expected_checksums, mismatch_counter, num_packet);
}
