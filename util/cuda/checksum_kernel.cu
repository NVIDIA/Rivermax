/*
 * SPDX-FileCopyrightText: Copyright (c) 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

static const int blockSize = 1024;

__global__ void 
cuda_compare_checksum_kernel(unsigned int expected, unsigned char* data,
                             unsigned int size, unsigned int* mismatches)
{
    int idx = threadIdx.x;

    // Calculate the sum for each thread.
    int sum = 0;
    for (int i = idx; i < size; i += blockSize)
        sum += data[i];

    __shared__ unsigned int accum[blockSize];
    accum[idx] = sum;

    // Reduce the sums of all blocks.
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) {
        if (idx < size)
            accum[idx] += accum[idx + size];
        __syncthreads();
    }

    // Output the results in the first thread.
    if (idx == 0 && accum[0] != expected)
        *mismatches = *mismatches + 1;
}

extern "C"
void cuda_compare_checksum(unsigned int expected, unsigned char* data,
                           unsigned int size, unsigned int* mismatches)
{
    cuda_compare_checksum_kernel<<<1, blockSize>>>(expected, data, size, mismatches);
}
